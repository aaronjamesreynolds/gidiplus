#include "hip/hip_runtime.h"
#include "Mini-app.cuh"

/*
=========================================================
*/
MCGIDI_HOST_DEVICE double LCG_random_double(uint64_t * seed)
{
	// LCG parameters
	const uint64_t m = 9223372036854775808ULL; // 2^63
	const uint64_t a = 2806196910506780709ULL;
	const uint64_t c = 1ULL;
	*seed = (a * (*seed) + c) % m;
	return (double) (*seed) / (double) m;
}	
       
/*
=========================================================
*/
MCGIDI_HOST_DEVICE uint64_t fast_forward_LCG(uint64_t seed, uint64_t n)
{
	// LCG parameters
	const uint64_t m = 9223372036854775808ULL; // 2^63
	uint64_t a = 2806196910506780709ULL;
	uint64_t c = 1ULL;

	n = n % m;

	uint64_t a_new = 1;
	uint64_t c_new = 0;

	while(n > 0) 
	{
		if(n & 1)
		{
			a_new *= a;
			c_new = c_new * a + c;
		}
		c *= (a + 1);
		a *= a;

		n >>= 1;
	}

	return (a_new * seed + c_new) % m;
}

/*
=========================================================
*/

bool approximatelyEqual(double a, double b, double epsilon)
{
    return (double) fabs(a - b) <= ( ((double) fabs(a) > (double) fabs(b) ? (double) fabs(b) : (double) fabs(a)) * epsilon);
}

/*
=========================================================
*/

bool approximatelyEqual(double *a,  double *b, int size, double epsilon)
{

  bool vectorsEqual = true;

  for (int i = 0; i < size; i++)
  {
    vectorsEqual = vectorsEqual and approximatelyEqual(a[i], b[i], epsilon);
  }

  return vectorsEqual;
}

/*
=========================================================
*/

double get_time()
{

       // If using C++, we can do this:
       unsigned long us_since_epoch = std::chrono::high_resolution_clock::now().time_since_epoch() / std::chrono::microseconds(1);
       return (double) us_since_epoch / 1.0e6;

}

/*
=========================================================
*/

void setCudaOptions()
{

    // Set and verify CUDA limits
    size_t my_size;
    hipDeviceSetLimit( hipLimitStackSize, 80 * 1024 );
    hipDeviceGetLimit( &my_size, hipLimitStackSize ) ;
    printf( "hipLimitStackSize =  %luk\n", my_size / 1024 );
    hipDeviceSetLimit( hipLimitMallocHeapSize, 100 * 1024 * 1024 );
    hipDeviceGetLimit( &my_size, hipLimitMallocHeapSize ) ;
    printf( "hipLimitMallocHeapSize =  %luM\n", my_size / ( 1024 * 1024 ) );
    hipDeviceSetLimit( hipLimitPrintfFifoSize, 40 * 1024 * 1024 );
    hipDeviceGetLimit( &my_size, hipLimitPrintfFifoSize );
    printf( "hipLimitPrintfFifoSize =  %luM\n", my_size / ( 1024 * 1024 ) );

}

/*
=========================================================
*/

