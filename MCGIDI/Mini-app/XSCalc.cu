#include "hip/hip_runtime.h"
/*
# <<BEGIN-copyright>>
# Copyright 2019, Lawrence Livermore National Security, LLC.
# See the top-level COPYRIGHT file for details.
# 
# SPDX-License-Identifier: MIT
# <<END-copyright>>
*/

/*
Brief: Cross section lookup kernels/functions
Author: Aaron James Reynolds 
*/

#include "Mini-app.cuh"

/*
===============================================================================
Perform macroscopic XS lookups on device
*/
template <typename T>
__global__ void calcTotalMacroXSs(
    char   ** deviceProtares,
    T      *  domainHash,
    int    *  materialComposition,
    double *  numberDensities,
    double *  verification,
    int       maxNumberIsotopes,
    int       numCollisions, 
    bool      sampleProduct) 
{       

  int collisionIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if( collisionIndex >= numCollisions ) return;

  // Data used to evaluate XS
  uint64_t                 seed           = STARTING_SEED;
  double                   temperature    = 2.58522e-8;
  MCGIDI::URR_protareInfos urr;

  // Fast-forward random number seed
  seed = fast_forward_LCG(seed, 2 * collisionIndex);
  
  // Sample material and energy
  double energy    = pow(10.0, LCG_random_double(&seed) * 1.3);
  int    matIndex  = pick_mat(&seed);
  int    hashIndex = domainHash->index(energy);
  
  // Initialize accumulators and loop variables
  double microCrossSection, totalCrossSection = 0, numberDensity = -1;
  int    isoIndex = -1;
  MCGIDI::ProtareSingle * MCProtare; 

  // Evaluate scattering and total XS
  for (int iConstituent = 0; 
       materialComposition[matIndex * maxNumberIsotopes + iConstituent] >= 0 
       && iConstituent < maxNumberIsotopes; 
       iConstituent++)
  {
    isoIndex      = materialComposition[matIndex * maxNumberIsotopes 
      + iConstituent];
    numberDensity = numberDensities[matIndex * maxNumberIsotopes 
      + iConstituent];

    MCProtare = reinterpret_cast<MCGIDI::ProtareSingle *>(deviceProtares[isoIndex]);

    microCrossSection  = MCProtare->crossSection(
        urr, 
        hashIndex, 
        temperature, 
        energy);
    totalCrossSection += numberDensity * microCrossSection;
  }

  // Sample reaction and product
  if (sampleProduct)
  {
    TallyProductHandler product = sampleProducts(
        MCProtare,
        hashIndex,
        temperature,
        energy, 
        microCrossSection,
        &seed);
  }

  verification[collisionIndex] = totalCrossSection;
}

/*
===============================================================================
Perform macroscopic XS lookups on host
*/
template <typename T>
void calcTotalMacroXSs(
    std::vector<MCGIDI::Protare *> protares,
    T                            * domainHash,
    int                          * materialComposition,
    double                       * numberDensities,
    double                       * verification,
    int                            maxNumberIsotopes,
    int                            numLookups,
    bool                           sampleProduct) 
{       

  // Data used to evaluate XS
  double                   temperature    = 2.58522e-8;
  MCGIDI::URR_protareInfos urr;

  for (int iXS = 0; iXS < numLookups; iXS++)
  {

    uint64_t seed = STARTING_SEED;
    seed = fast_forward_LCG(seed, 2 * iXS);

    // Sample material and energy
    double energy    = pow(10.0, LCG_random_double(&seed) * 1.3);
    int    matIndex  = pick_mat(&seed);
    int    hashIndex = domainHash->index(energy);

    // Initialize accumulators and loop variables
    double            microCrossSection, totalCrossSection = 0;
    double            numberDensity = -1;
    int               isoIndex = -1;
    MCGIDI::Protare * MCProtare; 

    // Evaluate scattering and total XS
    for (int iConstituent = 0; 
        materialComposition[matIndex * maxNumberIsotopes + iConstituent] >= 0 
        && iConstituent < maxNumberIsotopes; 
        iConstituent++)
    {
      isoIndex      = materialComposition[matIndex * maxNumberIsotopes 
        + iConstituent];
      numberDensity = numberDensities[matIndex * maxNumberIsotopes 
        + iConstituent];

      MCProtare = protares[isoIndex];

      microCrossSection  = MCProtare->crossSection(
          urr, hashIndex, temperature, energy);
      totalCrossSection += numberDensity * microCrossSection; 

    }

    // Sample reaction and product
    if (sampleProduct)
    {
      TallyProductHandler product = sampleProducts(
          MCProtare,
          hashIndex,
          temperature,
          energy, 
          microCrossSection,
          &seed);
    }

    verification[iXS] = totalCrossSection;

  }

}

/*
===============================================================================
Sample products given a protare and total cross section
*/
template <typename T>
MCGIDI_HOST_DEVICE TallyProductHandler sampleProducts(
    T        * MCProtare,
    int        hashIndex,
    double     temperature,
    double     energy, 
    double     crossSection,
    uint64_t * seed)
{

  // Declare sampling variables 
  MCGIDI::Sampling::Input  input( true, MCGIDI::Sampling::Upscatter::Model::B );
  TallyProductHandler      products;
  MCGIDI::URR_protareInfos urr;

  // Sample a reaction
  int reactionIndex = MCProtare->sampleReaction( 
      urr, 
      hashIndex, 
      temperature, 
      energy, 
      crossSection, 
      (double (*)(void *)) LCG_random_double, 
      &seed );
  MCGIDI::Reaction const *reaction = MCProtare->reaction( reactionIndex );

  // Sample products of reaction
  reaction->sampleProducts(
      MCProtare, 
      energy, 
      input, 
      (double (*)(void *)) LCG_random_double, 
      &seed, 
      products);

  return products;
}

// DEVICE INSTANTIATIONS

// CE XS lookup
template
__global__ void calcTotalMacroXSs(
    char               ** deviceProtares,
    MCGIDI::DomainHash  * domainHash,
    int                 * materialComposition,
    double              * numberDensities,
    double              * verification,
    int                   maxNumberIsotopes,
    int                   numCollisions,
    bool                  sampleProduct);

// MG XS lookup
template
__global__ void calcTotalMacroXSs(
    char                   ** deviceProtares,
    MCGIDI::MultiGroupHash  * domainHash,
    int                     * materialComposition,
    double                  * numberDensities,
    double                  * verification,
    int                       maxNumberIsotopes,
    int                       numCollision,
    bool                      sampleProduct);

// Sample products on host
template 
MCGIDI_HOST_DEVICE TallyProductHandler sampleProducts(
    MCGIDI::Protare * MCProtare,
    int               hashIndex,
    double            temperature,
    double            energy, 
    double            crossSection,
    uint64_t        * seed);

// HOST INSTANTIATIONS

// CE XS lookup
template 
void calcTotalMacroXSs(
    std::vector<MCGIDI::Protare *> protares,
    MCGIDI::DomainHash           * domainHash,
    int                          * materialComposition,
    double                       * numberDensities,
    double                       * verification,
    int                            maxNumberIsotopes,
    int                            numLookups,
    bool                           sampleProduct);

// MG XS lookup
template 
void calcTotalMacroXSs(
    std::vector<MCGIDI::Protare *> protares,
    MCGIDI::MultiGroupHash       * domainHash,
    int                          * materialComposition,
    double                       * numberDensities,
    double                       * verification,
    int                            maxNumberIsotopes,
    int                            numLookups,
    bool                           sampleProduct); 

// Sample products on host
template
MCGIDI_HOST_DEVICE TallyProductHandler sampleProducts(
    MCGIDI::ProtareSingle * MCProtare,
    int                     hashIndex,
    double                  temperature,
    double                  energy, 
    double                  crossSection,
    uint64_t              * seed);
