#include "hip/hip_runtime.h"
/*
# <<BEGIN-copyright>>
# Copyright 2019, Lawrence Livermore National Security, LLC.
# See the top-level COPYRIGHT file for details.
# 
# SPDX-License-Identifier: MIT
# <<END-copyright>>
*/

/*
Brief: Protare initialization and H-to-D memory operations
Author: Aaron James Reynolds 
(processes adapted from those in gpuTest.cpp)
*/

#include "Mini-app.cuh"

/*
===============================================================================
Serialize a protare (Taken from MCGICI/Test/gpuTest/gpuTest.cpp)

Call this each isotope per block and one warp only 
(i.e. <<< number_isotopes, 32>>>)
*/
__global__ void setUp( int a_numIsotopes, MCGIDI::DataBuffer **a_buf ) {  

    int isotopeIndex = blockIdx.x;

    MCGIDI::DataBuffer *buf = a_buf[isotopeIndex];
    MCGIDI::ProtareSingle *MCProtare = new(buf->m_placementStart) MCGIDI::ProtareSingle( );

    buf->zeroIndexes( );
    buf->m_placement = buf->m_placementStart + sizeof( MCGIDI::ProtareSingle );
    buf->m_maxPlacementSize = sizeof( *a_buf[isotopeIndex] ) + sizeof( MCGIDI::ProtareSingle );

    // This line causes a "nvlink warning".
    MCProtare->serialize( *buf, MCGIDI::DataBuffer::Mode::Unpack );                 
    buf->m_placement = buf->m_placementStart + sizeof( MCGIDI::ProtareSingle );
}

/*
===============================================================================
Initialize MG or CE protares for XS lookup rate calculation 
(Adapted from MCGICI/Test/gpuTest/gpuTest.cpp)

Using paths that were in gpuTest.cpp. Ideally, these paths would be defined in
the make file. 
*/
std::vector<MCGIDI::Protare *> initMCProtares(
    int numIsotopes, 
    const char *isotopeNames[], 
    energyMode mode, 
    int numHashBins)
{

    // Initialize protares and nuclear data maps
    std::vector<MCGIDI::Protare *>protares(numIsotopes);
    std::string mapFilename( "/usr/gapps/Mercury/data/nuclear/endl/2009.3_gp3.17/gnd/all.map" );
    PoPI::Database pops( "/usr/gapps/Mercury/data/nuclear/endl/2009.3/gnd/pops.xml" );
    std::ifstream meta_stream( "/usr/gapps/data/nuclear/development/GIDI3/Versions/V10/metastables_alias.xml" );
    std::string metastable_string( ( std::istreambuf_iterator<char>( meta_stream ) ), 
                                     std::istreambuf_iterator<char>( ) );
    pops.addDatabase( metastable_string, false );
    GIDI::Map::Map map( mapFilename, pops );
    MCGIDI::DomainHash domainHash( numHashBins, 1e-8, 10 );
    std::set<int>                  exclusionSet;

    // Initialize progress message
    std::string progress_str = "";

    // For each isotope referenced in isotopeNames, construct a GIDI::protare. 
    // Then, initialize a MCGIDI::protare from the GIDI::protare object. 
    double startTime = get_time();
    for( int isoIndex = 0; isoIndex < numIsotopes; isoIndex++ ) 
    {
      std::string protareFilename( map.protareFilename( PoPI::IDs::neutron, isotopeNames[isoIndex] ) );

      // Initialize GIDI::protare
      GIDI::Protare                * protare;
      GIDI::Construction::Settings   construction( 
          GIDI::Construction::ParseMode::excludeProductMatrices, 
          GIDI::Construction::PhotoMode::nuclearAndAtomic );
      protare = map.protare(
          construction, 
          pops, 
          PoPI::IDs::neutron, 
          isotopeNames[isoIndex] );

      // Initialize a CE or MG protare depending on mode
      // Note: only initializing MCGIDI protares with one temperature
      switch (mode)
      {
        case(ce):
          {
            GIDI::Styles::TemperatureInfos temperature = {protare->temperatures()[0]};
            std::string                    label( temperature[0].griddedCrossSection( ) );
            MCGIDI::Transporting::MC       MC( 
                pops, 
                PoPI::IDs::neutron, 
                &protare->styles( ), 
                label, 
                GIDI::Transporting::DelayedNeutrons::on, 
                20.0 );
            GIDI::Transporting::Particles  particleList;
            GIDI::Transporting::MultiGroup continuous_energy_multigroup;
            GIDI::Transporting::Particle   projectile( "n", continuous_energy_multigroup);
            particleList.add( projectile );

            // Construct MCGIDI::protare from GIDI::protare
            protares[isoIndex] = MCGIDI::protareFromGIDIProtare(
                *protare, 
                pops, 
                MC, 
                particleList, 
                domainHash, 
                temperature, 
                exclusionSet);
            break;
          }
        case(mg):
          {
            GIDI::Transporting::Particles  particleList;
            GIDI::Styles::TemperatureInfos temperature = {protare->temperatures()[0]};
            std::string label( temperature[0].heatedMultiGroup( ) );

            //GIDI::Transporting::Groups_from_bdfls groups_from_bdfls( "../../GIDI/Test/bdfls" );
            //GIDI::Transporting::Fluxes_from_bdfls fluxes_from_bdfls( "../../GIDI/Test/bdfls", 0.0 );
            GIDI::Transporting::Groups_from_bdfls groups_from_bdfls( "/collab/usr/gdata/nuclear/endl_official/endl2009.3/bdfls" );
            GIDI::Transporting::Fluxes_from_bdfls fluxes_from_bdfls( "/collab/usr/gdata/nuclear/endl_official/endl2009.3/bdfls", 0.0 );

            std::string gid( "LLNL_gid_7" );
            GIDI::Transporting::MultiGroup multi_group = groups_from_bdfls.viaLabel( gid );
            GIDI::Transporting::Particle projectile("n", multi_group );
            projectile.appendFlux( fluxes_from_bdfls.getViaFID( 1 ) );
            particleList.add( projectile );
            particleList.process( *protare, label );
            MCGIDI::Transporting::MC MC( 
                pops, 
                PoPI::IDs::neutron, 
                &protare->styles( ), 
                label, 
                GIDI::Transporting::DelayedNeutrons::on, 
                20.0 );
            MC.crossSectionLookupMode( MCGIDI::Transporting::LookupMode::Data1d::multiGroup );

            // Construct MCGIDI::protare from GIDI::protare
            protares[isoIndex] = MCGIDI::protareFromGIDIProtare( 
                *protare, 
                pops, 
                MC, 
                particleList, 
                domainHash, 
                temperature, 
                exclusionSet ); 
            break;
          }
      }

      // Clear out last progress message and print the new one
      std::cout << std::string(progress_str.length(),'\b');
      progress_str = "Initialized " + std::to_string(isoIndex) 
        + " / " + std::to_string(numIsotopes) + " protares";
      std::cout << progress_str;

    }
    double endTime = get_time();
    double elapsedTime = endTime - startTime;

    // Print protare intialization time
    std::cout << std::string(progress_str.length(),'\b');
    printf("Initialized %d MCGIDI protares in %f seconds.\n", numIsotopes, elapsedTime);

    return protares;

}

/*
===============================================================================
Print reaction data for a list of protares 
(Taken from MCGICI/Test/gpuTest/gpuTest.cpp)
*/
void printReactionData(std::vector<MCGIDI::Protare *> protares)
{

  int numIsotopes = protares.size();

  // For the each  protare, print out the possible reactions and their thresholds
  for( int isoIndex = 0; isoIndex < numIsotopes; isoIndex++ ) 
  {

    MCGIDI::Protare *MCProtare = protares[isoIndex];
    int numberOfReactions = MCProtare->numberOfReactions( );
    MCGIDI::Sampling::Input input( true, MCGIDI::Sampling::Upscatter::Model::B );
    MCGIDI::Sampling::MCGIDIVectorProductHandler products;

    for( int iReaction = 0; iReaction < numberOfReactions; ++iReaction ) 
    {
      MCGIDI::Reaction const *reaction = MCProtare->reaction( iReaction );
      double                 threshold = MCProtare->threshold( iReaction );

      printf( "HO: reaction(%d) = %s threshold = %g ENDF_MT = %d\n" , 
          iReaction, reaction->label( ).c_str( ), threshold, reaction->ENDF_MT());
    }
  }

}

/*
===============================================================================
Copy protares from host to device (Taken from MCGICI/Test/gpuTest/gpuTest.cpp)
*/
std::vector<char *> copyProtaresFromHostToDevice(
    std::vector<MCGIDI::Protare *> protares)
{

  int numIsotopes = protares.size();
  size_t totalSize = 0;

  // Build data buffer to copy host MCGIDI::protares to device
  std::vector<MCGIDI::DataBuffer *>deviceBuffers_h( numIsotopes );
  std::vector<char *>deviceProtares( numIsotopes );
  for( int isoIndex = 0; isoIndex < numIsotopes; isoIndex++ ) 
  {
    MCGIDI::DataBuffer buf_h;

    protares[isoIndex]->serialize( buf_h, MCGIDI::DataBuffer::Mode::Count );

    buf_h.allocateBuffers( );
    buf_h.zeroIndexes( );
    protares[isoIndex]->serialize( buf_h, MCGIDI::DataBuffer::Mode::Pack );

    size_t cpuSize = protares[isoIndex]->memorySize( );
    totalSize += cpuSize;
    deviceBuffers_h[isoIndex] = buf_h.copyToDevice( cpuSize, deviceProtares[isoIndex] );
  }

  // Copy data buffer from host to device
  MCGIDI::DataBuffer **deviceBuffers_d = nullptr;
  hipMalloc( (void **) &deviceBuffers_d, sizeof( MCGIDI::DataBuffer * ) * numIsotopes );
  hipMemcpy( deviceBuffers_d, &deviceBuffers_h[0], sizeof( MCGIDI::DataBuffer * ) * numIsotopes, hipMemcpyHostToDevice );

  printf("Copied %d buffered MCGIDI protares (%d MB) from host to device.\n", 
      numIsotopes,
      int(totalSize / (1024.0 * 1024.0)));

  setUp<<< numIsotopes, 32 >>>( numIsotopes, deviceBuffers_d );

  gpuErrchk( hipPeekAtLastError( ) );
  gpuErrchk( hipDeviceSynchronize( ) );

  return deviceProtares;

}

/*
===============================================================================
Initialize a multigroup hash and return a pointer to it
*/
MCGIDI::MultiGroupHash * getMGHash(const char *isotopeNames[])
{

  // Initialize protares and nuclear data maps
  std::vector<MCGIDI::Protare *> protares(1);
  std::string mapFilename( "/usr/gapps/Mercury/data/nuclear/endl/2009.3_gp3.17/gnd/all.map" );
  PoPI::Database pops( "/usr/gapps/Mercury/data/nuclear/endl/2009.3/gnd/pops.xml" );
  std::ifstream meta_stream( "/usr/gapps/data/nuclear/development/GIDI3/Versions/V10/metastables_alias.xml" );
  std::string metastable_string( ( std::istreambuf_iterator<char>( meta_stream ) ), 
      std::istreambuf_iterator<char>( ) );
  pops.addDatabase( metastable_string, false );
  GIDI::Map::Map map( mapFilename, pops );
  MCGIDI::DomainHash domainHash( 4000, 1e-8, 10 );

  std::string protareFilename( map.protareFilename( PoPI::IDs::neutron, isotopeNames[0] ) );

  // Initialize GIDI::protare
  GIDI::Protare                *protare;
  GIDI::Construction::Settings construction( 
      GIDI::Construction::ParseMode::excludeProductMatrices, 
      GIDI::Construction::PhotoMode::nuclearAndAtomic );
  protare = map.protare( construction, pops, PoPI::IDs::neutron, isotopeNames[0] );

  GIDI::Styles::TemperatureInfos temperatures = protare->temperatures( );
  MCGIDI::MultiGroupHash * mgDomainHash = new MCGIDI::MultiGroupHash( *protare, temperatures[0] );

  return mgDomainHash;
}

/*
===============================================================================
Initialize a continuous energy hash and return a pointer to it
*/
MCGIDI::DomainHash * getCEHash(const int nBins)
{

  MCGIDI::DomainHash * ceDomainHash = new MCGIDI::DomainHash(nBins, 1e-8, 10 );

  return ceDomainHash;

}

