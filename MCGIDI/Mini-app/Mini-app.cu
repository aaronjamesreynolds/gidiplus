#include "Mini-app.cuh"

/*
=========================================================
*/
int main( int argc, char **argv ) 
{

    Input in = Input(argc, argv);

    // Print logo
    in.printLogo();

    // Print runtime options
    printf("=== RUNTIME OPTIONS ===\n\n");
    in.printInputOptions();

#if defined(_OPENMP)
    // Set number of OMP threads for CPU hash calc
    omp_set_num_threads(in.numOMPThreads);
#endif

    printf("=== INITIALIZING PROTARES ===\n\n");
    
    // Set up material compositions and number densities
    std::vector<std::vector<int>> materialCompositions2D = initMaterialCompositions(in.problem_size);
    std::vector<std::vector<double>> numberDensities2D = initNumberDensities(materialCompositions2D);

    int    numMats = materialCompositions2D.size(), 
           maxNumIsotopes = materialCompositions2D[0].size(),
           numEntries = numMats * maxNumIsotopes;
    int    *materialCompositions;
    double *numberDensities;
    double *verification;

    int deviceId;
    hipGetDevice(&deviceId);

    // Allocate memory for material data
    size_t sizeMatComp      = numEntries * sizeof(int);
    size_t sizeNumDens      = numEntries * sizeof(double);
    size_t sizeVerification = in.numLookups * sizeof(double);
    hipMallocManaged(&materialCompositions, sizeMatComp);
    hipMallocManaged(&numberDensities,      sizeNumDens); 
    hipMallocManaged(&verification,         sizeVerification); 

    // Initialize 1D material composition and number density vectors
    unwrapFrom2Dto1D(materialCompositions2D, materialCompositions, numMats, maxNumIsotopes);
    unwrapFrom2Dto1D(numberDensities2D, numberDensities, numMats, maxNumIsotopes);

    // Copy material compositions and number densities to device
    hipMemPrefetchAsync(materialCompositions, sizeMatComp,      deviceId);
    hipMemPrefetchAsync(numberDensities,      sizeNumDens,      deviceId);
    hipMemPrefetchAsync(verification,         sizeVerification, deviceId);

    // Set and verify CUDA limits
    // These options were in gpuTest. If I use them, I run out of device memory, so I'm not using them.
    //setCudaOptions();

    // Initialize protares and nuclear data maps
    std::vector<MCGIDI::Protare *> protares = initMCProtares(in.numIsotopes, in.isotopeNames);

    // Print reaction data
    if (in.printData) printReactionData(protares);
  
    // Serialize protares, then copy them from host to device
    std::vector<char *> deviceProtares = copyProtaresFromHostToDevice(protares);

    // Calculate number of blocks in execution configuration
    int numBlocks = (in.numLookups + in.numThreads - 1) / in.numThreads;

    printf("\n=== XS CALCULATION ===\n\n");

    printf("TOTAL XS\n");
    printf("========\n");
    printf("Calculating total XSs on GPU...\n");

    // Launch and time macroscopic total XS sampling kernel 
    double startTime = get_time();
    for (int iBatch = 0; iBatch < in.numBatches; iBatch++)
    {
      calcTotalMacroXSs<<<numBlocks, in.numThreads>>>(
          &deviceProtares[0], 
          materialCompositions, 
          numberDensities,
          verification,
          maxNumIsotopes,
          in.numLookups);
      gpuErrchk( hipPeekAtLastError( ) );
      gpuErrchk( hipDeviceSynchronize( ) );
    }
    double endTime  = get_time();

    // Get XS calculation rate
    double elapsedTime = endTime - startTime;
    double xs_rate = (double) in.numBatches * in.numLookups / elapsedTime;

    // Print out look-up rate
    printf("Looked up %d * %g XSs in %g seconds \n", in.numBatches, static_cast<double>(in.numLookups), elapsedTime);
    printf("Total XS look-up rate: %g cross sections per second \n\n", xs_rate);

    if (in.numToVerify > 0)
    {
      uint64_t seed = STARTING_SEED;
      int verifyStart = LCG_random_double(&seed) * (in.numLookups - in.numToVerify);
    
      printf("Calculating total XSs on CPU and verifying consistency... \n");

      // Get CPU comparison hash
      startTime = get_time();
      bool verification_match = calcTotalMacroXSs(
          protares,
          materialCompositions,
          numberDensities,
          verification,
          maxNumIsotopes,
          verifyStart,
          in.numToVerify);
      endTime  = get_time();
      elapsedTime = endTime - startTime;

      printf("CPU verification completed in %g seconds.\n\n", elapsedTime);
      if (verification_match)
        printf("Success! GPU and CPU total XSs for lookups %d through %d match!.\n\n",verifyStart, verifyStart + in.numToVerify - 1);
      else
      {
        printf("Failure! GPU and CPU total XSs for lookups %d through %d DO NOT match!.\n\n",verifyStart, verifyStart + in.numToVerify - 1);
      }
    }
    else
      printf("To verify consistency between host and device execution, set doCompare = 1.\n\n");

    printf("SCATTER XS\n");
    printf("==========\n");

    printf("Calculating scatter XSs on GPU... \n");

    // Launch and time macroscopic scattering XS sampling kernel 
    startTime = get_time();
    for (int iBatch = 0; iBatch < in.numBatches; iBatch++)
    {
      calcScatterMacroXSs<<<numBlocks, in.numThreads>>>(
          &deviceProtares[0], 
          materialCompositions, 
          numberDensities,
          verification,
          maxNumIsotopes,
          in.numLookups);
      gpuErrchk( hipPeekAtLastError( ) );
      gpuErrchk( hipDeviceSynchronize( ) );
    }
    endTime  = get_time();
    
    // Get XS calculation rate
    elapsedTime = endTime - startTime;
    xs_rate = (double) in.numBatches * in.numLookups / elapsedTime;

    // Print out look-up rate
    printf("Looked up %d * %g XSs in %g seconds \n", in.numBatches, static_cast<double>(in.numLookups), elapsedTime);
    printf("Scatter XS look-up rate: %g cross sections per second \n\n", xs_rate);

    if (in.numToVerify > 0)
    {
      uint64_t seed = STARTING_SEED;
      int verifyStart = LCG_random_double(&seed) * (in.numLookups - in.numToVerify);
    
      printf("Calculating scatter XSs on CPU... \n");

      // Get CPU comparison hash
      startTime = get_time();
      bool verification_match = calcScatterMacroXSs(
          protares,
          materialCompositions,
          numberDensities,
          verification,
          maxNumIsotopes,
          verifyStart,
          in.numToVerify);
      endTime  = get_time();
      elapsedTime = endTime - startTime;

      printf("CPU verification completed in %g seconds.\n\n", elapsedTime);
      if (verification_match)
        printf("Success! GPU and CPU scatter XSs for lookups %d through %d match!.\n\n",verifyStart, verifyStart + in.numToVerify - 1);
      else
      {
        printf("Failure! GPU and CPU scatter XSs for lookups %d through %d DO NOT match!.\n\n",verifyStart, verifyStart + in.numToVerify - 1);
      }
    }
    else
      printf("To verify consistency between host and device execution, set doCompare = 1.\n\n");

    return( EXIT_SUCCESS );
}


